/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static __global__ void fractal(const int width, const int start_frame, const int gpu_frames, unsigned char* const pic)
{
  // todo: use the GPU to compute the requested frames (base the code on the previous project)


  const double Delta = 0.002;
  const double xMid = 0.2315059;
  const double yMid = 0.5214880;

  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < (gpu_frames - start_frame)* width * width)
    {
    //frames
    const int frame = i / (width * width);
    double delta = Delta * pow(0.98,frame);

    const double xMin = xMid - delta;
    const double yMin = yMid - delta;
    const double dw = 2.0 * delta / width;

    const int row = (i / width) % width;  // rows
      const double cy = yMin + row * dw;

    const int col = i % width;  // columns
        const double cx = xMin + col * dw;
        double x = cx;
        double y = cy;
        double x2, y2;
        int depth = 256;
        do {
          x2 = x * x;
          y2 = y * y;
          y = 2.0 * x * y + cy;
          x = x2 - y2 + cx;
          depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));
        //pic[frame * width * width + row * width + col] = (unsigned char)depth;
	  pic[i] = (unsigned char)depth;
  }
}

unsigned char* GPU_Init(const int gpu_frames, const int width){
  unsigned char* d_pic;
  if (hipSuccess != hipMalloc((void **)&d_pic, gpu_frames * width * width * sizeof(unsigned char))) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  return d_pic;
}

void GPU_Exec(const int start_frame, const int gpu_frames, const int width, unsigned char* d_pic)
{
  // todo: launch the kernel with ThreadsPerBlock and the appropriate number of blocks (do not wait for the kernel to finish)

    fractal<<<((gpu_frames * width * width ) + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(width, start_frame, gpu_frames, d_pic);
}

void GPU_Fini(const int gpu_frames, const int width, unsigned char* pic, unsigned char* d_pic)
{
  // todo: copy the result from the device to the host and free the device memory

    if (hipSuccess != hipMemcpy(pic, d_pic, sizeof(unsigned char) * gpu_frames * width * width, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); exit(-1);}

    hipFree(d_pic);
}

