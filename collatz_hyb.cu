/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static int* d_maxlen;

static __global__ void collatz(const long start, const long stop, int* const maxlen)
{
  // todo: process odd values from start (assume start to be odd) to stop (inclusively if stop is odd) with one thread per value (based on code from previous project)

   const long i = start + 2*(threadIdx.x + blockIdx.x * (long) blockDim.x);

 if(i <= stop){
    long val = i;
    int len = 1;
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val = val / 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }

    if(len > *maxlen)
        atomicMax(maxlen, len);
 }
}

void GPU_Init()
{
  int maxlen = 0;
  if (hipSuccess != hipMalloc((void **)&d_maxlen, sizeof(int))) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  if (hipSuccess != hipMemcpy(d_maxlen, &maxlen, sizeof(int), hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n"); exit(-1);}
}

void GPU_Exec(const long start, const long stop)
{
  if (start <= stop) {
    collatz<<<((stop - start + 2) / 2 + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(start, stop, d_maxlen);
  }
}

int GPU_Fini()
{
  int maxlen;

  // todo: copy the result from the device to the host and free the device memory

  if (hipSuccess != hipMemcpy(&maxlen, d_maxlen, sizeof(int), hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); exit(-1);}
  hipFree(d_maxlen);

  return maxlen;
}

